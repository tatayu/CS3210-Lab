#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <ctype.h>
#include <errno.h>
#include "settings.h"

extern "C" {
#include "goi_cuda.h"
}

#define LINUX
long long wall_clock_time()
{
#ifdef LINUX
    struct timespec tp;
    clock_gettime(CLOCK_REALTIME, &tp);
    return (long long)(tp.tv_nsec + (long long)tp.tv_sec * 1000000000ll);
#else
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (long long)(tv.tv_usec * 1000 + (long long)tv.tv_sec * 1000000000ll);
#endif
}

// including the "dead faction": 0
#define MAX_FACTIONS 10

// this macro is here to make the code slightly more readable, not because it can be safely changed to
// any integer value; changing this to a non-zero value may break the code
#define DEAD_FACTION 0

void check_cuda_errors()
{
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        printf("Last CUDA error %s\n", hipGetErrorString(rc));
    }

}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a dead cell to become alive.
 */
 __host__ __device__ bool isBirthable(int n)
{
    return n == 3;
}

/**
 * Specifies the number(s) of live neighbors of the same faction required for a live cell to remain alive.
 */
 __host__ __device__ bool isSurvivable(int n)
{
    return n == 2 || n == 3;
}

/**
 * Specifies the number of live neighbors of a different faction required for a live cell to die due to fighting.
 */
 __host__ __device__ bool willFight(int n) {
    return n > 0;
}

/**
 * returns the value at the input row and col of the input grid, if valid.
 * 
 * -1 is returned if row or col is out of bounds (as specified by nRows and nCols).
 */
 __host__ __device__ int getValueAt(const int *grid, int nRows, int nCols, int row, int col)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return -1;
    }

    return *(grid + (row * nCols) + col);
}

/**
 * sets the value at the input row and col of the input grid to val.
 * 
 * Does nothing if row or col is out of bounds (as specified by nRows and nCols).
 */
 
 __host__ __device__ void setValueAt(int *grid, int nRows, int nCols, int row, int col, int val)
{
    if (row < 0 || row >= nRows || col < 0 || col >= nCols)
    {
        return;
    }

    *(grid + (row * nCols) + col) = val;
}

/**
 * Computes and returns the next state of the cell specified by row and col based on currWorld and invaders. Sets *diedDueToFighting to
 * true if this cell should count towards the death toll due to fighting.
 * 
 * invaders can be NULL if there are no invaders.
 */
__host__ __device__ int getNextState(const int *currWorld, const int *invaders, int nRows, int nCols, int row, int col, bool *diedDueToFighting)
{
    // we'll explicitly set if it was death due to fighting
    *diedDueToFighting = false;

    // faction of this cell
    int cellFaction = getValueAt(currWorld, nRows, nCols, row, col);

    // did someone just get landed on?
    if (invaders != NULL && getValueAt(invaders, nRows, nCols, row, col) != DEAD_FACTION)
    {   
        *diedDueToFighting = cellFaction != DEAD_FACTION;
        return getValueAt(invaders, nRows, nCols, row, col);
    }

    // tracks count of each faction adjacent to this cell
    int neighborCounts[MAX_FACTIONS];
    memset(neighborCounts, 0, MAX_FACTIONS * sizeof(int));

    // count neighbors (and self)
    for (int dy = -1; dy <= 1; dy++)
    {
        for (int dx = -1; dx <= 1; dx++)
        {
            int faction = getValueAt(currWorld, nRows, nCols, row + dy, col + dx);
            if (faction >= DEAD_FACTION)
            {
                neighborCounts[faction]++;
            }
        }
    }

    // we counted this cell as its "neighbor"; adjust for this
    neighborCounts[cellFaction]--;

    if (cellFaction == DEAD_FACTION)
    {
        // this is a dead cell; we need to see if a birth is possible:
        // need exactly 3 of a single faction; we don't care about other factions

        // by default, no birth
        int newFaction = DEAD_FACTION;

        // start at 1 because we ignore dead neighbors
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            int count = neighborCounts[faction];
            if (isBirthable(count))
            {
                newFaction = faction;
            }
        }

        return newFaction;
    }
    else
    {
        /** 
         * this is a live cell; we follow the usual rules:
         * Death (fighting): > 0 hostile neighbor
         * Death (underpopulation): < 2 friendly neighbors and 0 hostile neighbors
         * Death (overpopulation): > 3 friendly neighbors and 0 hostile neighbors
         * Survival: 2 or 3 friendly neighbors and 0 hostile neighbors
         */

        int hostileCount = 0;
        for (int faction = DEAD_FACTION + 1; faction < MAX_FACTIONS; faction++)
        {
            if (faction == cellFaction)
            {
                continue;
            }
            hostileCount += neighborCounts[faction];
        }

        if (willFight(hostileCount))
        {
            *diedDueToFighting = true;
            return DEAD_FACTION;
        }

        int friendlyCount = neighborCounts[cellFaction];
        if (!isSurvivable(friendlyCount))
        {
            return DEAD_FACTION;
        }

        return cellFaction;
    }
}

__device__ __managed__ int deathToll = 0;
//__device__ __managed__ int counter = 0;

__global__ void updateState(int nRows, int nCols, int *world, int *inv, int *wholeNewWorld, int nTask, int rowPerThread, int remainder)
{
    //atomicAdd(&counter, 1);
    //printf("counter: %d\n", counter);
    int blkid = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int tid = blkid * (blockDim.x*blockDim.y*blockDim.z) + (threadIdx.z*(blockDim.x*blockDim.y))
    + (threadIdx.y*blockDim.x) + threadIdx.x;
    
    if(tid < nTask)
    {
        int start = rowPerThread * tid + (tid >= remainder ? remainder : tid);
        if(tid < remainder)
        {
            rowPerThread += 1;
        }

        for(int row = start; row < start + rowPerThread; row ++)
        {
            for(int col = 0; col < nCols; col ++)
            {
                bool diedDueToFighting;
                int nextState = getNextState(world, inv, nRows, nCols, row, col, &diedDueToFighting);
                setValueAt(wholeNewWorld, nRows, nCols, row, col, nextState); 
                if (diedDueToFighting)
                {
                    atomicAdd(&deathToll, 1); //!global, mutex
                    //("deathToll added: %d\n", deathToll);
                }
            }
        }
    }
    else
    {
        return;
    }
}
/**
 * The main simulation logic.
 * 
 * goi does not own startWorld, invasionTimes or invasionPlans and should not modify or attempt to free them.
 * nThreads is the number of threads to simulate with. It is ignored by the sequential implementation.
 */
int goi(int gridX, int gridY, int gridZ, int blockX, int blockY, int blockZ, int nGenerations, const int *startWorld, int nRows, int nCols, int nInvasions, const int *invasionTimes, int **invasionPlans)
{
    //!CLOCK
    long long before, after;
    int *gpu_world = NULL;
    hipError_t rc;

    before = wall_clock_time();
    //!!!!
    // death toll due to fighting
    deathToll = 0;
    
    int nThread = gridX * gridY * gridZ * blockX * blockY * blockZ;
    int nTask = nRows * nCols;
    int rowPerThread = nRows / nThread;
    int remainder = nRows % nThread;

    dim3 dimGrd(gridX, gridY, gridZ);
    dim3 dimBlk(blockX, blockY, blockZ);
    
    hipMalloc((void **)&gpu_world, sizeof(int) * nRows * nCols);
    rc = hipMemcpy(gpu_world, startWorld, sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
    if (rc != hipSuccess)
    {
        printf("Could not copy to device for world. Reason: %s\n", hipGetErrorString(rc));
    }

    if (gpu_world == NULL)
    {
        //free(world);
        hipFree(gpu_world);
        return -1;
    }

    int *gpu_wholeNewWorld = NULL;
    hipMalloc((void **)&gpu_wholeNewWorld, sizeof(int) * nRows * nCols);
    if (gpu_wholeNewWorld == NULL)
    {
        hipFree(gpu_world);
        return -1;
    }

    int *temp = NULL;
    hipMalloc((void **)&temp, sizeof(int) * nRows * nCols);
   
    // Begin simulating
    int invasionIndex = 0;
    for (int i = 1; i <= nGenerations; i++)
    {
        // is there an invasion this generation?
        int *gpu_inv = NULL;
        if (invasionIndex < nInvasions && i == invasionTimes[invasionIndex])
        {
            // we make a copy because we do not own invasionPlans
            hipMalloc((void **)&gpu_inv, sizeof(int) * nRows * nCols);
            rc = hipMemcpy(gpu_inv, invasionPlans[invasionIndex], sizeof(int) * nRows * nCols, hipMemcpyHostToDevice);
            if (rc != hipSuccess)
            {
                printf("Could not copy to device for world. Reason: %s\n", hipGetErrorString(rc));
            }

            if (gpu_inv == NULL)
            {
                hipFree(gpu_inv);
                return -1;
            }
            invasionIndex++;
        }
        
        //************************************************CUDA CODE************************************************
        //int *gpu_wholeNewWorld = NULL;
        //hipMalloc((void **)&gpu_wholeNewWorld, sizeof(int) * nRows * nCols);
        updateState<<<dimGrd, dimBlk>>>(nRows, nCols, gpu_world, gpu_inv, gpu_wholeNewWorld, nTask, rowPerThread, remainder);
        check_cuda_errors();
        hipDeviceSynchronize();
        //*********************************************************************************************************

        if (gpu_inv != NULL)
        {
            hipFree(gpu_inv);
        }

        // swap worlds
        temp = gpu_world;
        gpu_world = gpu_wholeNewWorld;
        gpu_wholeNewWorld = temp;
    }

    //free(world);
    hipFree(temp);
    hipFree(gpu_world);
    hipFree(gpu_wholeNewWorld);

    //!clock end
    after = wall_clock_time();
    fprintf(stderr, "Operation took %1.2f seconds\n", ((float)(after - before)) / 1000000000);
    //!

    return deathToll;
}
